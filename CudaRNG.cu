#include <stdio.h>
#include <stdlib.h>
#include <dirent.h>
#include <hip/hip_runtime.h>
#include <string.h>
#include <sys/stat.h>
#include <sys/types.h>
#include <unistd.h>
#include <cmath>
#include <errno.h>


#define Cm hipMalloc
#define Cmc hipMemcpy
#define CmcHD hipMemcpyHostToDevice
#define CmcDH hipMemcpyDeviceToHost


__host__ void cpuBinConverter(int temprature,int* fin_bin)
{
    for(int i=31;i>=0;i--)
    {
        int x=((int)temprature>>i)&1;
        fin_bin[31-i]=x;
    }
}

__global__ void gpuBinConverter(int d_temprature,int* d_fin_bin)
{
    int tid=threadIdx.x+blockDim.x*blockIdx.x;
    __shared__ int hold;
    if(tid<32)
    {
        int hold=((int)d_temprature>>tid)&1;
        d_fin_bin[31-tid]=hold;
    }
}

__host__ void cpuIntConverter(int* cells,unsigned int &ret,int size)
{
    int hold;
    for(int i=0;i<size;i++)
    {
        hold+=pow(2,size-i-1)*cells[i];
    }
    ret=hold;
}

__global__ void gpuIntConverter(int* d_cells,unsigned int *d_ret,int d_size)
{
    int tid=threadIdx.x+blockDim.x*blockIdx.x;
    __shared__ int hold;
    if(tid==0)
    {
        hold=0;
    }
    if(tid<d_size)
    {
        atomicAdd(&hold,pow(2,d_size-tid-1)*d_cells[tid]);
    }
    __syncthreads();
    if(tid==0)
    {
        *d_ret=hold;

    }
}

__global__ void moveRight(int* arr,int* res,int delta)
{
    int value=arr[threadIdx.x];
    value=__shfl_up_sync(0xFFFFFFFF,value,delta);
    res[threadIdx.x]=value;
    res[0]=0;
}

__global__ void hillisSteeleScan(int* arr,int *temp,int size)
{

    int tid=threadIdx.x+blockDim.x*blockIdx.x;
    temp[tid]=arr[tid];
    __syncthreads();
    for(int stride=1;stride<=(size)/2;stride*=2)
    {
        if(tid>=stride)
        {
            temp[tid]=arr[tid]+arr[tid-stride];
        }
        else
        {
            temp[tid]=arr[tid];
        }
        __syncthreads();

        arr[tid]=temp[tid];

        __syncthreads();
    }
}   

__host__ void cpuTransf(int* cells,int* out, int size)
{
    for(int i=0;i<size;i++)
    {
        out[i]=cells[i];
    }
}

__global__ void gpuTransf(int* d_cells,int* d_out,int d_size)
{
    int tid=threadIdx.x+blockDim.x*blockIdx.x;
    if(tid<d_size)
    {
        atomicAdd(&d_out[tid],d_cells[tid]);
    }
    __syncthreads();
}

__host__ void elementaryCellGenerate(int* cells,int* out,int size,int* rule)
{
    int p,q,r;
    for(int i=1;i<size-1;i++)
    {
        p=4*cells[i-1];
        q=2*cells[i];
        r=cells[i+1];
        out[i]=rule[7-(p+q+r)];
    }
}

__global__ void gpuElementaryCellularAutoRule30(int* d_cells,int* d_out, int d_size,int* d_rule)
{
    int tid=threadIdx.x+blockDim.x*blockIdx.x;
    __shared__ int p,q,r;
    __shared__ int hold;
    if(tid>0 && tid<d_size-1)
    {
        p=4*d_cells[tid-1];
        q=2*d_cells[tid];
        r=d_cells[tid+1];
        hold=d_rule[7-(p+q+r)];
        d_out[tid]=hold;
    }
}   

__host__ void cpuPrinter(int* cells,int size)
{
    for(int i=0;i<size;i++)
    {
        if(cells[i]==1) printf("1");
        else printf("0");
    }
    printf("\n");
}

__global__ void gpuPrinter(int *d_cells,int d_size)
{
    for(int i=0;i<d_size;i++)
    {
        if(d_cells[i]==1) printf("1");
        else printf("0");
    }
    printf("\n");
}

__host__ void cudaRNG(int seed,int size,int* rule,int blocksize)
{
    int *cells=(int*)malloc(size*sizeof(int));
    unsigned int ret;
    unsigned int *d_ret;
    int *d_cells;
    int *d_res;
    int *d_out;
    int *d_rule;

    cpuBinConverter(seed,cells);
    cpuPrinter(cells,size);

    Cm((int**)&d_cells,size*sizeof(int));
    Cm((int**)&d_res,size*sizeof(int));//temporary variables
    Cm((int**)&d_out,size*sizeof(int));//temporary variables 
    Cm((void**)&d_ret,sizeof(unsigned int));
    Cm((int**)&d_rule,8*sizeof(int));

    Cmc(d_cells,cells,size*sizeof(int),CmcHD);
    Cmc(d_rule,rule,8*sizeof(int),CmcHD);

    int no_blocks=(size+blocksize-1)/blocksize;
    for(int i=0;i<(size/2);i++)
    {
        gpuPrinter<<<1,1>>>(d_cells,size);
        gpuElementaryCellularAutoRule30<<<no_blocks,blocksize>>>(d_cells,d_out,size,d_rule);

        hipDeviceSynchronize();

        gpuTransf<<<no_blocks,blocksize>>>(d_out,d_cells,size);
    }
    gpuIntConverter<<<1,32>>>(d_cells,d_ret,size);

    hipDeviceSynchronize();

    Cmc(&ret,d_ret,sizeof(unsigned int),CmcDH);
    printf("%d",ret);
}

int main(int argc, char** argv) 
{
    char path[256];
    char filePath[512];
    DIR *dir;
    FILE *file;
    float temperature;
    float fin_temp;
    struct dirent *ent;
    
    strcpy(path,"/sys/class/hwmon/");

    dir=opendir(path);
    if(dir==NULL)
    {
        perror("opendir");
        return 1;
    }

    while ((ent = readdir(dir)) != NULL) {
        if (strcmp(ent->d_name, ".") == 0 || strcmp(ent->d_name, "..") == 0)
            continue;
        snprintf(filePath,sizeof(filePath),"%s%s",path,ent->d_name);

        DIR *subdir = opendir(filePath);
        if (subdir==NULL) 
        {
            perror("opendir subdir");
            continue;
        }

        struct dirent *subent;
        while ((subent = readdir(subdir)) != NULL) {
            if (strstr(subent->d_name, "temp") && strstr(subent->d_name, "_input")) {
                snprintf(filePath,sizeof(filePath),"%s/%s/%s",path,ent->d_name,subent->d_name);

                file = fopen(filePath,"r");
                if (file==NULL) 
                {
                    perror("fopen");
                    continue;
                }

                fscanf(file, "%f", &temperature);
                printf("%2f \n",temperature/1000.0);
                fin_temp=temperature/1000.0;
                fclose(file);
            }
        }
        closedir(subdir);
    }
    closedir(dir);

    int rule[8]={0,0,0,1,1,1,1,0};
    int out[64]={0};
    unsigned int r;

    //gpu generation implemenatation
    printf("The GPU implementation: \n");
    cudaRNG((int)fin_temp,64,rule,32);
    printf("\n");

    int *d_temp;
    int *d_cells;
    
    //cpu generation for refernce
    printf("The CPU implementation: \n");
    // for(int i=0;i<32;i++)
    // {
    //     cpuPrinter(cells,64);
    //     elementaryCellGenerate(cells,out,64,rule);
    //     cpuTransf(out,cells,64);
    // }
    // cpuPrinter(out,64);
    // cpuIntConverter(out,r,64);
    // printf("The random number generated is %d", r);

    return 0;
}
